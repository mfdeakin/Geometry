#include "hip/hip_runtime.h"

#include "cudadef.h"

#include "vector.hpp"

#include <random>

#include <gtest/gtest.h>

constexpr static const int dim = 3;
using fptype = float;

using V = Geometry::Vector<dim, fptype>;

TEST(Vector, CudaCopy) {
	constexpr const fptype minValue = -(1 << 30);
	constexpr const fptype maxValue = 1 << 30;
	std::random_device rd;
	std::mt19937_64 rng(rd());
	std::uniform_real_distribution<fptype> pdf(minValue, maxValue);
	constexpr const int numTests = 1000;
	for(int t = 0; t < numTests; t++) {
		V vSrc;
		for(int i = 0; i < dim; i++) {
			fptype coord = pdf(rng);
			vSrc.set(i, coord);
		}
		{
			std::shared_ptr<V::VectorData> ptr1 = vSrc.cudaCopy();
			V vDest1;
			vDest1.cudaRetrieve(ptr1);
			ASSERT_EQ(vDest1, vSrc);
			V vDest2;
			vDest2.cudaRetrieve(ptr1.get());
			ASSERT_EQ(vDest2, vSrc);
		}
		{
			V::VectorData *ptr2 = NULL;
			hipMalloc(&ptr2, sizeof(*ptr2));
			vSrc.cudaCopy(ptr2);
			V vDest3;
			vDest3.cudaRetrieve(ptr2);
			ASSERT_EQ(vDest3, vSrc);
			hipFree(ptr2);
		}
		{
			V::VectorData *ptr3 = NULL;
			hipMalloc(&ptr3, sizeof(*ptr3));
			std::shared_ptr<V::VectorData> sptr3(ptr3, hipFree);
			vSrc.cudaCopy(sptr3);
			V vDest4;
			vDest4.cudaRetrieve(sptr3);
			ASSERT_EQ(vDest4, vSrc);
			V vDest5;
			vDest5.cudaRetrieve(sptr3.get());
			ASSERT_EQ(vDest5, vSrc);
		}
	}
}

__global__ void CudaAdd(V::VectorData *lhs, V::VectorData *rhs,
                        V::VectorData *dest, int numVectors) {
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	const int numThreads = blockDim.x * gridDim.x;
	const int vpt = numVectors / numThreads + 1;
	const int threadIdx = idx * vpt;
	for(int i = 0; i < vpt && threadIdx + i < numVectors; i++) {
		V vLHS(lhs[threadIdx + i]);
		V vRHS(rhs[threadIdx + i]);
		V sum = vLHS + vRHS;
		sum.copy(&dest[threadIdx + i]);
	}
}

TEST(Vector, CudaAdd) {
	constexpr const fptype minValue = -(1 << 30);
	constexpr const fptype maxValue = 1 << 30;
	std::random_device rd;
	std::mt19937_64 rng(rd());
	std::uniform_real_distribution<fptype> pdf(minValue, maxValue);
	constexpr const int numVecs = 100;
	constexpr const int numTests = 100;
	for(int t = 0; t < numTests; t++) {
		V lhs[numVecs];
		V rhs[numVecs];
		V::VectorData *lhsMem, *rhsMem, *destMem;
		hipMalloc(&lhsMem, sizeof(V::VectorData[numVecs]));
		hipMalloc(&rhsMem, sizeof(V::VectorData[numVecs]));
		hipMalloc(&destMem, sizeof(V::VectorData[numVecs]));
		for(int i = 0; i < numVecs; i++) {
			for(int j = 0; j < dim; j++) {
				lhs[i].set(j, pdf(rng));
				rhs[i].set(j, pdf(rng));
			}
			lhs[i].cudaCopy(&lhsMem[i]);
			rhs[i].cudaCopy(&rhsMem[i]);
		}
		constexpr const int numBlocks = 32;
		constexpr const int numThreads = 32;
		CudaAdd<<<numBlocks, numThreads>>>(lhsMem, rhsMem, destMem, numVecs);
		for(int i = 0; i < numVecs; i++) {
			V sum = lhs[i] + rhs[i];
			V cmp;
			cmp.cudaRetrieve(&destMem[i]);
			ASSERT_EQ(sum, cmp);
		}
	}
}
